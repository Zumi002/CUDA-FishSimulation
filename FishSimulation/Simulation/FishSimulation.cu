#include "FishSimulation.h"

int FishSimulation::getMaxFishCount()
{
	return maxFishCount;
}

int FishSimulation::getFishCount()
{
	return fishCount;
}

void FishSimulation::setUpSimulation(FishVBOs* fishVBOs)
{
	vbos = fishVBOs;
	fishCount = 0;
	fishData = FishData();
	allocFishTypes();
	addFishType(FishType());
	mapVBOs();
}

void FishSimulation::mapVBOs()
{

	hipGraphicsResource* crPosX,
		* crPosY,
		* crVelX,
		* crVelY;

	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosX, vbos->posXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosY, vbos->posYVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelX, vbos->velXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelY, vbos->velYVBO, hipGraphicsRegisterFlagsNone));

	// Map all resources once
	gpuErrchk(hipGraphicsMapResources(1, &crPosX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosX, nullptr, crPosX));

	gpuErrchk(hipGraphicsMapResources(1, &crPosY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosY, nullptr, crPosY));

	gpuErrchk(hipGraphicsMapResources(1, &crVelX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelX, nullptr, crVelX));

	gpuErrchk(hipGraphicsMapResources(1, &crVelY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelY, nullptr, crVelY));
	gpuErrchk(hipMalloc((void**)&fishData.type, maxFishCount * sizeof(short)));
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMemset(fishData.devTempVelX, 0, maxFishCount * sizeof(float)));
	gpuErrchk(hipMemset(fishData.devTempVelY, 0, maxFishCount * sizeof(float)));
}

void FishSimulation::allocFishTypes()
{
	fishTypes = new FishTypes();

	fishTypes->alignRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coheherentRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separateRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->alignFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coherentFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separationFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->obstacleAvoidanceFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->maxSpeed = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->minSpeed = (float*)malloc(maxFishTypes * sizeof(float));

	devfishTypes = FishTypes();
	gpuErrchk(hipMalloc(&devfishTypes.alignRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coheherentRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separateRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.alignFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coherentFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separationFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.obstacleAvoidanceFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.maxSpeed, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.minSpeed, maxFishTypes * sizeof(float)));
}

void FishSimulation::syncFishTypes()
{
	gpuErrchk(hipMemcpy(devfishTypes.alignRange, fishTypes->alignRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coheherentRange, fishTypes->coheherentRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separateRange, fishTypes->separateRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.alignFactor, fishTypes->alignFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coherentFactor, fishTypes->coherentFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separationFactor, fishTypes->separationFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.obstacleAvoidanceFactor, fishTypes->obstacleAvoidanceFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.maxSpeed, fishTypes->maxSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.minSpeed, fishTypes->minSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
}

void FishSimulation::addFishType(FishType fishType)
{
	fishTypes->alignRange[fishTypesCount] = fishType.alignRange;
	fishTypes->coheherentRange[fishTypesCount] = fishType.coheherentRange;
	fishTypes->separateRange[fishTypesCount] = fishType.separateRange;
	fishTypes->alignFactor[fishTypesCount] = fishType.alignFactor;
	fishTypes->coherentFactor[fishTypesCount] = fishType.coherentFactor;
	fishTypes->separationFactor[fishTypesCount] = fishType.separationFactor;
	fishTypes->obstacleAvoidanceFactor[fishTypesCount] = fishType.obstacleAvoidanceFactor;
	fishTypes->maxSpeed[fishTypesCount] = fishType.maxSpeed;
	fishTypes->minSpeed[fishTypesCount] = fishType.minSpeed;

	fishTypesCount++;
}

int FishSimulation::calcBlocksNeeded(int amount, int threadsCount)
{
	return (amount + threadsCount - 1) / threadsCount;
}

void FishSimulation::addFish(int amount, short type)
{
	int blocks = calcBlocksNeeded(amount, 512);

	setFishTypeKernel << <blocks, 512 >> > (fishData, type, amount, fishCount);
	hipDeviceSynchronize();
	randomizePos(amount, fishCount);
	fishCount += amount;
	
}

int FishSimulation::getFishTypeCount()
{
	return fishTypesCount;
}

FishTypes* FishSimulation::getFishTypes()
{
	return fishTypes;
}

MousePos* FishSimulation::getMousePos()
{
	return mousePos;
}

void FishSimulation::randomizePos(int count, int offset)
{
	int blocks = calcBlocksNeeded(count, 512);

	randomizePositionKernel << <blocks, 512 >> > (fishData, count, offset);
	hipDeviceSynchronize();
}

void FishSimulation::simulationStep()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	simulateStepKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount, *mousePos);
	hipDeviceSynchronize();
	updatePositionKernel << <blocks, 512 >> > (fishData, fishCount);
	hipDeviceSynchronize();
}
