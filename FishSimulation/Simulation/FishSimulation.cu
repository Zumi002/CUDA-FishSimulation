#include "hip/hip_runtime.h"
#include "FishSimulation.h"

int FishSimulation::getMaxFishCount()
{
	return maxFishCount;
}

int FishSimulation::getFishCount()
{
	return fishCount;
}

void FishSimulation::setUpSimulation(FishVBOs* fishVBOs)
{
	vbos = fishVBOs;
	fishCount = 0;
	fishData = FishData();
	allocFishTypes();
	addFishType(FishType());
	mapVBOs();
	allocTempFishData();
}

void FishSimulation::mapVBOs()
{

	

	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosX, vbos->posXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosY, vbos->posYVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelX, vbos->velXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelY, vbos->velYVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crColor, vbos->colorVBO, hipGraphicsRegisterFlagsNone));

	// Map all resources once
	gpuErrchk(hipGraphicsMapResources(1, &crPosX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosX, nullptr, crPosX));

	gpuErrchk(hipGraphicsMapResources(1, &crPosY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosY, nullptr, crPosY));

	gpuErrchk(hipGraphicsMapResources(1, &crVelX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelX, nullptr, crVelX));

	gpuErrchk(hipGraphicsMapResources(1, &crVelY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelY, nullptr, crVelY));

	gpuErrchk(hipGraphicsMapResources(1, &crColor, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devColorRGBA, nullptr, crColor));

	
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&fishData.devType, maxFishCount * sizeof(short)));
	gpuErrchk(hipMalloc((void**)&fishData.devID, maxFishCount * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&fishData.devGridCell, maxFishCount * sizeof(int)));
	gpuErrchk(hipMemset(fishData.devTempVelX, 0, maxFishCount * sizeof(float)));
	gpuErrchk(hipMemset(fishData.devTempVelY, 0, maxFishCount * sizeof(float)));
}
void FishSimulation::allocTempFishData()
{
	gpuErrchk(hipMalloc((void**)&tempFishData.devPosX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devPosY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devVelX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devVelY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devTempVelX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devTempVelY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devType, maxFishCount * sizeof(short)))
	gpuErrchk(hipMalloc((void**)&tempFishData.devColorRGBA, maxFishCount * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devID, maxFishCount * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&tempFishData.devGridCell, maxFishCount * sizeof(int)));
	//no need for allocating color
	//no need for allocating ID
	//no need for allocating gridCell
	
}

void FishSimulation::allocFishTypes()
{
	fishTypes = new FishTypes();

	fishTypes->alignRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coherentRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separateRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->alignFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coherentFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separateFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->obstacleAvoidanceFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->maxSpeed = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->minSpeed = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->color = (int*)malloc(maxFishTypes * sizeof(int));

	devfishTypes = FishTypes();
	gpuErrchk(hipMalloc(&devfishTypes.alignRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coherentRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separateRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.alignFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coherentFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separateFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.obstacleAvoidanceFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.maxSpeed, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.minSpeed, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.color, maxFishTypes * sizeof(float)));
}

void FishSimulation::syncFishTypes()
{
	gpuErrchk(hipMemcpy(devfishTypes.alignRange, fishTypes->alignRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coherentRange, fishTypes->coherentRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separateRange, fishTypes->separateRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.alignFactor, fishTypes->alignFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coherentFactor, fishTypes->coherentFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separateFactor, fishTypes->separateFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.obstacleAvoidanceFactor, fishTypes->obstacleAvoidanceFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.maxSpeed, fishTypes->maxSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.minSpeed, fishTypes->minSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.color, fishTypes->color, fishTypesCount * sizeof(int), hipMemcpyHostToDevice));
}

void FishSimulation::addFishType(FishType fishType)
{
	if (fishTypesCount < maxFishTypes)
	{
		fishTypes->alignRange[fishTypesCount] = fishType.alignRange;
		fishTypes->coherentRange[fishTypesCount] = fishType.coherentRange;
		fishTypes->separateRange[fishTypesCount] = fishType.separateRange;
		fishTypes->alignFactor[fishTypesCount] = fishType.alignFactor;
		fishTypes->coherentFactor[fishTypesCount] = fishType.coherentFactor;
		fishTypes->separateFactor[fishTypesCount] = fishType.separateFactor;
		fishTypes->obstacleAvoidanceFactor[fishTypesCount] = fishType.obstacleAvoidanceFactor;
		fishTypes->maxSpeed[fishTypesCount] = fishType.maxSpeed;
		fishTypes->minSpeed[fishTypesCount] = fishType.minSpeed;
		fishTypes->color[fishTypesCount] = fishType.color;

		fishTypesCount++;
	}
}

int FishSimulation::calcBlocksNeeded(int amount, int threadsCount)
{
	return (amount + threadsCount - 1) / threadsCount;
}

void FishSimulation::addFish(int amount, short type)
{
	if (fishCount < maxFishCount)
	{
		amount = std::min(amount, maxFishCount - fishCount);
		int blocks = calcBlocksNeeded(amount, 512);

		setFishTypeKernel << <blocks, 512 >> > (fishData, type, amount, fishCount);
		gpuErrchk(hipDeviceSynchronize());
		randomizePos(amount, fishCount);
		fishCount += amount;
	}
}

int FishSimulation::getFishTypeCount()
{
	return fishTypesCount;
}

FishTypes* FishSimulation::getFishTypes()
{
	return fishTypes;
}

MousePos* FishSimulation::getMousePos()
{
	return mousePos;
}

void FishSimulation::randomizePos(int count, int offset)
{
	int blocks = calcBlocksNeeded(count, 512);

	randomizePositionKernel << <blocks, 512 >> > (fishData, devfishTypes, count, offset);
	gpuErrchk(hipDeviceSynchronize());
}

void FishSimulation::makeGrid()
{
	float gridSize = 1;
	int blocks = calcBlocksNeeded(fishCount, 512);
	for (int i = 0; i < fishTypesCount; i++)
	{
		gridSize = fmaxf((*fishTypes).alignRange[i], gridSize);
		gridSize = fmaxf((*fishTypes).separateRange[i], gridSize);
		gridSize = fmaxf((*fishTypes).coherentRange[i], gridSize);
	}
	collumns = ceilf((float)2*1000 / gridSize);
	int rows = ceilf((float)2*1000 / gridSize); // 2* world size
	cellCount = collumns * rows; 
	if (devGridStart != nullptr)
	{
		gpuErrchk(hipFree(devGridStart));
	}
	gpuErrchk(hipMalloc(&devGridStart, cellCount*sizeof(int)));
	gpuErrchk(hipMemset(devGridStart, -1, cellCount * sizeof(int)));
	preGridMakingKernel << <blocks, 512 >> > (fishData, tempFishData, fishCount, devGridStart, gridSize, cellCount, collumns);
	gpuErrchk(hipDeviceSynchronize());
	auto devPointerGridCell = thrust::device_pointer_cast(fishData.devGridCell);
	auto devPointerDevID = thrust::device_pointer_cast(fishData.devID);
	thrust::sort_by_key(devPointerGridCell, devPointerGridCell + fishCount, devPointerDevID);
	gpuErrchk(hipDeviceSynchronize());
	postGridMakingKernel << <blocks, 512 >> > (fishData, tempFishData, fishCount, devGridStart);
	gpuErrchk(hipDeviceSynchronize());
}

void FishSimulation::simulationStep()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	simulateStepKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount, *mousePos);
	gpuErrchk(hipDeviceSynchronize());
	updatePositionKernel << <blocks, 512 >> > (fishData, fishCount);
	gpuErrchk(hipDeviceSynchronize());
}

void FishSimulation::simulationStepGrid()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	makeGrid();
	simulateStepGridKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount, *mousePos, devGridStart, cellCount, collumns);
	gpuErrchk(hipDeviceSynchronize());
	updatePositionKernel << <blocks, 512 >> > (fishData, fishCount);
	gpuErrchk(hipDeviceSynchronize());
}

void FishSimulation::pauseInteractions()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	pauseInteractionsKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount);
	gpuErrchk(hipDeviceSynchronize());
}

void FishSimulation::cleanUp()
{
	unmapVBOs();
	freeFishTypes();
	delete fishTypes;
}

void FishSimulation::unmapVBOs()
{
	gpuErrchk(hipGraphicsUnmapResources(1, &crPosX, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crPosY, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crVelX, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crVelY, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crColor, 0));

	// Unregister VBOs
	gpuErrchk(hipGraphicsUnregisterResource(crPosX));
	gpuErrchk(hipGraphicsUnregisterResource(crPosY));
	gpuErrchk(hipGraphicsUnregisterResource(crVelX));
	gpuErrchk(hipGraphicsUnregisterResource(crVelY));
	gpuErrchk(hipGraphicsUnregisterResource(crColor));

	//hipFree remaining buffers
	gpuErrchk(hipFree(fishData.devType));
	gpuErrchk(hipFree(fishData.devTempVelX));
	gpuErrchk(hipFree(fishData.devTempVelY));
	gpuErrchk(hipFree(fishData.devID));
	gpuErrchk(hipFree(fishData.devGridCell));
}

void FishSimulation::freeFishTypes()
{

	//free host fishTypes
	delete[] fishTypes->alignRange;
	delete[] fishTypes->coherentRange;
	delete[] fishTypes->separateRange;
	delete[] fishTypes->alignFactor;
	delete[] fishTypes->coherentFactor;
	delete[] fishTypes->separateFactor;
	delete[] fishTypes->obstacleAvoidanceFactor;
	delete[] fishTypes->minSpeed;
	delete[] fishTypes->maxSpeed;
	delete[] fishTypes->color;

	//free device fishTypes
	gpuErrchk(hipFree(devfishTypes.alignRange));
	gpuErrchk(hipFree(devfishTypes.coherentRange));
	gpuErrchk(hipFree(devfishTypes.separateRange));
	gpuErrchk(hipFree(devfishTypes.alignFactor));
	gpuErrchk(hipFree(devfishTypes.coherentFactor));
	gpuErrchk(hipFree(devfishTypes.separateFactor));
	gpuErrchk(hipFree(devfishTypes.obstacleAvoidanceFactor));
	gpuErrchk(hipFree(devfishTypes.minSpeed));
	gpuErrchk(hipFree(devfishTypes.maxSpeed));
	gpuErrchk(hipFree(devfishTypes.color));
}

FishSimulation::~FishSimulation()
{
	cleanUp();
}
