#include "hip/hip_runtime.h"
#include "FishSimulation.h"

int FishSimulation::getMaxFishCount()
{
	return maxFishCount;
}

int FishSimulation::getFishCount()
{
	return fishCount;
}

void FishSimulation::setUpSimulation(FishVBOs* fishVBOs)
{
	vbos = fishVBOs;
	fishCount = 0;
	fishData = FishData();
	allocFishTypes();
	addFishType(FishType());
	mapVBOs();
}

void FishSimulation::mapVBOs()
{

	

	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosX, vbos->posXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crPosY, vbos->posYVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelX, vbos->velXVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crVelY, vbos->velYVBO, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&crColor, vbos->colorVBO, hipGraphicsRegisterFlagsNone));

	// Map all resources once
	gpuErrchk(hipGraphicsMapResources(1, &crPosX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosX, nullptr, crPosX));

	gpuErrchk(hipGraphicsMapResources(1, &crPosY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devPosY, nullptr, crPosY));

	gpuErrchk(hipGraphicsMapResources(1, &crVelX, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelX, nullptr, crVelX));

	gpuErrchk(hipGraphicsMapResources(1, &crVelY, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devVelY, nullptr, crVelY));

	gpuErrchk(hipGraphicsMapResources(1, &crColor, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&fishData.devColorRGBA, nullptr, crColor));

	gpuErrchk(hipMalloc((void**)&fishData.type, maxFishCount * sizeof(short)));
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelX, maxFishCount * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&fishData.devTempVelY, maxFishCount * sizeof(float)));
	gpuErrchk(hipMemset(fishData.devTempVelX, 0, maxFishCount * sizeof(float)));
	gpuErrchk(hipMemset(fishData.devTempVelY, 0, maxFishCount * sizeof(float)));
}

void FishSimulation::allocFishTypes()
{
	fishTypes = new FishTypes();

	fishTypes->alignRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coherentRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separateRange = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->alignFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->coherentFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->separateFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->obstacleAvoidanceFactor = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->maxSpeed = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->minSpeed = (float*)malloc(maxFishTypes * sizeof(float));
	fishTypes->color = (int*)malloc(maxFishTypes * sizeof(int));

	devfishTypes = FishTypes();
	gpuErrchk(hipMalloc(&devfishTypes.alignRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coherentRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separateRange, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.alignFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.coherentFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.separateFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.obstacleAvoidanceFactor, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.maxSpeed, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.minSpeed, maxFishTypes * sizeof(float)));
	gpuErrchk(hipMalloc(&devfishTypes.color, maxFishTypes * sizeof(float)));
}

void FishSimulation::syncFishTypes()
{
	gpuErrchk(hipMemcpy(devfishTypes.alignRange, fishTypes->alignRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coherentRange, fishTypes->coherentRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separateRange, fishTypes->separateRange, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.alignFactor, fishTypes->alignFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.coherentFactor, fishTypes->coherentFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.separateFactor, fishTypes->separateFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.obstacleAvoidanceFactor, fishTypes->obstacleAvoidanceFactor, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.maxSpeed, fishTypes->maxSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.minSpeed, fishTypes->minSpeed, fishTypesCount * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devfishTypes.color, fishTypes->color, fishTypesCount * sizeof(int), hipMemcpyHostToDevice));
}

void FishSimulation::addFishType(FishType fishType)
{
	if (fishTypesCount < maxFishTypes)
	{
		fishTypes->alignRange[fishTypesCount] = fishType.alignRange;
		fishTypes->coherentRange[fishTypesCount] = fishType.coherentRange;
		fishTypes->separateRange[fishTypesCount] = fishType.separateRange;
		fishTypes->alignFactor[fishTypesCount] = fishType.alignFactor;
		fishTypes->coherentFactor[fishTypesCount] = fishType.coherentFactor;
		fishTypes->separateFactor[fishTypesCount] = fishType.separateFactor;
		fishTypes->obstacleAvoidanceFactor[fishTypesCount] = fishType.obstacleAvoidanceFactor;
		fishTypes->maxSpeed[fishTypesCount] = fishType.maxSpeed;
		fishTypes->minSpeed[fishTypesCount] = fishType.minSpeed;
		fishTypes->color[fishTypesCount] = fishType.color;

		fishTypesCount++;
	}
}

int FishSimulation::calcBlocksNeeded(int amount, int threadsCount)
{
	return (amount + threadsCount - 1) / threadsCount;
}

void FishSimulation::addFish(int amount, short type)
{
	if (fishCount < maxFishCount)
	{
		amount = std::min(amount, maxFishCount - fishCount);
		int blocks = calcBlocksNeeded(amount, 512);

		setFishTypeKernel << <blocks, 512 >> > (fishData, type, amount, fishCount);
		hipDeviceSynchronize();
		randomizePos(amount, fishCount);
		fishCount += amount;
	}
}

int FishSimulation::getFishTypeCount()
{
	return fishTypesCount;
}

FishTypes* FishSimulation::getFishTypes()
{
	return fishTypes;
}

MousePos* FishSimulation::getMousePos()
{
	return mousePos;
}

void FishSimulation::randomizePos(int count, int offset)
{
	int blocks = calcBlocksNeeded(count, 512);

	randomizePositionKernel << <blocks, 512 >> > (fishData, devfishTypes, count, offset);
	hipDeviceSynchronize();
}

void FishSimulation::simulationStep()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	simulateStepKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount, *mousePos);
	hipDeviceSynchronize();
	updatePositionKernel << <blocks, 512 >> > (fishData, fishCount);
	hipDeviceSynchronize();
}

void FishSimulation::pauseInteractions()
{
	int blocks = calcBlocksNeeded(fishCount, 512);
	syncFishTypes();
	pauseInteractionsKernel << <blocks, 512 >> > (fishData, devfishTypes, fishCount);
	hipDeviceSynchronize();
}

void FishSimulation::cleanUp()
{
	unmapVBOs();
	freeFishTypes();
	delete fishTypes;
}

void FishSimulation::unmapVBOs()
{
	gpuErrchk(hipGraphicsUnmapResources(1, &crPosX, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crPosY, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crVelX, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crVelY, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &crColor, 0));

	// Unregister VBOs
	gpuErrchk(hipGraphicsUnregisterResource(crPosX));
	gpuErrchk(hipGraphicsUnregisterResource(crPosY));
	gpuErrchk(hipGraphicsUnregisterResource(crVelX));
	gpuErrchk(hipGraphicsUnregisterResource(crVelY));
	gpuErrchk(hipGraphicsUnregisterResource(crColor));

	//hipFree remaining buffers
	gpuErrchk(hipFree(fishData.type));
	gpuErrchk(hipFree(fishData.devTempVelX));
	gpuErrchk(hipFree(fishData.devTempVelY));
}

void FishSimulation::freeFishTypes()
{

	//free host fishTypes
	delete[] fishTypes->alignRange;
	delete[] fishTypes->coherentRange;
	delete[] fishTypes->separateRange;
	delete[] fishTypes->alignFactor;
	delete[] fishTypes->coherentFactor;
	delete[] fishTypes->separateFactor;
	delete[] fishTypes->obstacleAvoidanceFactor;
	delete[] fishTypes->minSpeed;
	delete[] fishTypes->maxSpeed;
	delete[] fishTypes->color;

	//free device fishTypes
	gpuErrchk(hipFree(devfishTypes.alignRange));
	gpuErrchk(hipFree(devfishTypes.coherentRange));
	gpuErrchk(hipFree(devfishTypes.separateRange));
	gpuErrchk(hipFree(devfishTypes.alignFactor));
	gpuErrchk(hipFree(devfishTypes.coherentFactor));
	gpuErrchk(hipFree(devfishTypes.separateFactor));
	gpuErrchk(hipFree(devfishTypes.obstacleAvoidanceFactor));
	gpuErrchk(hipFree(devfishTypes.minSpeed));
	gpuErrchk(hipFree(devfishTypes.maxSpeed));
	gpuErrchk(hipFree(devfishTypes.color));
}

FishSimulation::~FishSimulation()
{
	cleanUp();
}
