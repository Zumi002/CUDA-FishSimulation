#include "hip/hip_runtime.h"
#include "FishKernels.h"


__global__ void randomizePositionKernel(FishData fd, FishTypes ft, int count, int offset)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= count)
		return;
	//generate random values
	hiprandState state;
	hiprand_init(RANDSEED, idx, offset, &state);
	float randomX = hiprand_uniform(&state);

	float randomY = hiprand_uniform(&state);

	float randomVX = hiprand_uniform(&state) * 2 - 1;
	float randomVY = hiprand_uniform(&state) * 2 - 1;

	//scale values
	randomX = randomX * 2000 - 1000;
	randomY = randomY * 2000 - 1000;

	//apply value
	fd.devPosX[idx + offset] = randomX;
	fd.devPosY[idx + offset] = randomY;
	fd.devVelX[idx + offset] = randomVX;
	fd.devVelY[idx + offset] = randomVY;

	short type = fd.devType[idx + offset];
	fd.devColorRGBA[idx + offset] = ft.color[type];
}

__global__ void setFishTypeKernel(FishData fd, short type, int count, int offset)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= count)
		return;

	fd.devType[idx + offset] = type;
}

__global__ void simulateStepKernel(FishData fd, FishTypes ft, int fishCount, MousePos mousePos)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	float posx = fd.devPosX[idx];
	float posy = fd.devPosY[idx];
	float vx = fd.devVelX[idx];
	float vy = fd.devVelY[idx];
	short type = fd.devType[idx];
	float sepRangesq = ft.separateRange[type] * ft.separateRange[type];
	float alignSq = ft.alignRange[type] * ft.alignRange[type];
	float cohSq = ft.coherentRange[type] * ft.coherentRange[type];
	float sepFactor = ft.separateFactor[type];
	float cohFactor = ft.coherentFactor[type];
	float alignFactor = ft.alignFactor[type];
	float obstAvFactor = ft.obstacleAvoidanceFactor[type];

	Speed2D alignSpeed = Speed2D(),
		cohSpeed = Speed2D(),
		sepSpeed = Speed2D(),
		obstAvSpeed = Speed2D();

	float yposavg = 0;
	float xposavg = 0;

	int cohNeigh = 0;
	bool alignNeigh = 0,
		sepNeigh = 0,
		avoidingObst = 0;

	for (int i = 0; i < fishCount; i++)
	{
		if (i == idx)
			continue;

		float otherx = fd.devPosX[i];
		float othery = fd.devPosY[i];
		float dx = posx - otherx;
		float dy = posy - othery;


		float distsq = dx * dx + dy * dy;

		if (distsq <= sepRangesq)
		{
			float dist = sqrt(distsq);
			float d = 1 / (fmaxf(dist, 0.000001f)); //closer fish, bigger force
			sepSpeed.vx += dx*d;
			sepSpeed.vy += dy*d;

			sepNeigh = true;
		}

		if (distsq <= alignSq)
		{
			alignSpeed.vx += fd.devVelX[i];
			alignSpeed.vy += fd.devVelY[i];

			alignNeigh = true;
		}

		if (distsq <= cohSq)
		{
			xposavg += otherx;
			yposavg += othery;

			cohNeigh++;
		}
	}

	if (cohNeigh > 0)
	{
		xposavg /= (float)cohNeigh;
		yposavg /= (float)cohNeigh;

		xposavg -= posx;
		yposavg -= posy;

		cohSpeed = Speed2D(xposavg, yposavg);
	}


	if (mousePos.avoid)
	{
		float mdx = posx - mousePos.x;
		float mdy = posy - mousePos.y;

		float dist = mdy * mdy + mdx * mdx;

		if (dist < 40000)
		{
			obstAvSpeed.vx += mdx;
			obstAvSpeed.vy += mdy;
			avoidingObst = true;
		}
	}

	float minSpeed = ft.minSpeed[type];
	float maxSpeed = ft.maxSpeed[type];

	alignSpeed = steerTowards(alignSpeed, vx, vy, maxSpeed, 0.2f);
	cohSpeed = steerTowards(cohSpeed, vx, vy, maxSpeed, 0.2f);
	sepSpeed = steerTowards(sepSpeed, vx, vy, maxSpeed, 0.2f);
	obstAvSpeed = steerTowards(obstAvSpeed, vx, vy, maxSpeed, 0.2f);

	Speed2D sum = Speed2D(vx, vy);

	//we add steering only when needed 
	if (alignNeigh)
		sum.addScaled(alignSpeed, alignFactor);
	if (cohNeigh)
		sum.addScaled(cohSpeed, cohFactor);
	if (sepNeigh)
		sum.addScaled(sepSpeed, sepFactor);
	if (avoidingObst)
		sum.addScaled(obstAvSpeed, obstAvFactor);

	sum.max(maxSpeed);
	sum.min(minSpeed);

	fd.devColorRGBA[idx] = ft.color[type];

	fd.devTempVelX[idx] = sum.vx;
	fd.devTempVelY[idx] = sum.vy;
}

__global__ void updatePositionKernel(FishData fd, int fishCount)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	float vx = fd.devTempVelX[idx];
	float vy = fd.devTempVelY[idx];

	float posx = fd.devPosX[idx] + vx;
	float posy = fd.devPosY[idx] + vy;


	if (posx > 1000 || posx < -1000)
	{
		vx = -vx;
	}
	if (posy > 1000 || posy < -1000)
	{
		vy = -vy;
	}

	fd.devVelX[idx] = vx;
	fd.devVelY[idx] = vy;

	fd.devPosX[idx] = clamp(posx, -1000, 1000);
	fd.devPosY[idx] = clamp(posy, -1000, 1000);

	//printf("%f %f %f %f %f %f %d\n", fd.devTempVelX[idx], fd.devTempVelY[idx], fd.devVelX[idx], fd.devVelY[idx], fd.devPosX[idx], fd.devPosY[idx], idx);
}

__device__ float clamp(float value, float minVal, float maxVal)
{
	return fminf(fmaxf(value, minVal), maxVal);
}

__device__ Speed2D capSpeed(Speed2D speed2d, float minSpeed, float maxSpeed)
{
	float vx = speed2d.vx;
	float vy = speed2d.vy;

	float speed = sqrt(vx * vx + vy * vy);
	if (speed > 0)
	{
		vx /= speed;
		vy /= speed;
	}

	speed = clamp(speed, minSpeed, maxSpeed);

	return Speed2D(vx * speed, vy * speed);
}

__device__ Speed2D steerTowards(Speed2D speed2d, float vx, float vy, float maxSpeed, float steeringForce)
{
	speed2d.setMag(maxSpeed);
	speed2d.vx -= vx;
	speed2d.vy -= vy;
	speed2d.max(steeringForce);

	return speed2d;
}

__global__ void pauseInteractionsKernel(FishData fd, FishTypes ft, int fishCount)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	fd.devColorRGBA[idx] = ft.color[fd.devType[idx]];
}

__global__ void preGridMakingKernel(FishData fd, FishData tempFd, int fishCount, float gridSize, int cellCount, int collumns)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	float posX = fd.devPosX[idx];
	float posY = fd.devPosY[idx];
	tempFd.devPosX[idx] = posX;
	tempFd.devPosY[idx] = posY;
	tempFd.devVelX[idx] = fd.devVelX[idx];
	tempFd.devVelY[idx] = fd.devVelY[idx];
	tempFd.devType[idx] = fd.devType[idx];
	

	posX += 1000.0f;
	posY += 1000.0f;

	int row = ceilf(fminf(posY,1999) / gridSize)-1;
	int collumn = ceilf(fminf(posX, 1999) / gridSize)-1;
	if (row <= -1)
		row = 0;
	if (collumn <= -1)
		collumn = 0;
	int cell = row * collumns + collumn;

	fd.devGridCell[idx] = cell;
	fd.devID[idx] = idx;
	tempFd.devGridCell[idx] = cell;
	
	//printf("%d %d %d %d %f %f\n",idx, cell, row, collumn, posX, posY);

}

__global__ void postGridMakingKernel(FishData fd, FishData tempFd, int fishCount, int* gridStarts, int* gridEnds)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	int tmpIdx = fd.devID[idx];

	fd.devPosX[idx] = tempFd.devPosX[tmpIdx];
	fd.devPosY[idx] = tempFd.devPosY[tmpIdx];
	fd.devVelX[idx] = tempFd.devVelX[tmpIdx];
	fd.devVelY[idx] = tempFd.devVelY[tmpIdx];
	fd.devType[idx] = tempFd.devType[tmpIdx];
	fd.devGridCell[idx] = tempFd.devGridCell[tmpIdx];

	if (idx == 0)
	{
		gridStarts[fd.devGridCell[idx]] = idx;
	}
	else if (idx == fishCount - 1)
	{
		gridEnds[fd.devGridCell[idx]] = fishCount;
	}
	else if (fd.devGridCell[idx - 1] != fd.devGridCell[idx])
	{
		gridEnds[fd.devGridCell[idx - 1]] = idx;
		gridStarts[fd.devGridCell[idx]] = idx;
	}
}

__global__ void simulateStepGridKernel(FishData fd, FishTypes ft, int fishCount, MousePos mousePos, int* gridStarts, int* gridEnds, int cellCount, int collumns)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= fishCount)
		return;

	float posx = fd.devPosX[idx];
	float posy = fd.devPosY[idx];
	float vx = fd.devVelX[idx];
	float vy = fd.devVelY[idx];
	short type = fd.devType[idx];
	float sepRangesq = ft.separateRange[type] * ft.separateRange[type];
	float alignSq = ft.alignRange[type] * ft.alignRange[type];
	float cohSq = ft.coherentRange[type] * ft.coherentRange[type];
	float sepFactor = ft.separateFactor[type];
	float cohFactor = ft.coherentFactor[type];
	float alignFactor = ft.alignFactor[type];
	float obstAvFactor = ft.obstacleAvoidanceFactor[type];

	Speed2D alignSpeed = Speed2D(),
		cohSpeed = Speed2D(),
		sepSpeed = Speed2D(),
		obstAvSpeed = Speed2D();

	float yposavg = 0;
	float xposavg = 0;

	int cohNeigh = 0;
	bool alignNeigh = 0,
		sepNeigh = 0,
		avoidingObst = 0;

	int myCell = fd.devGridCell[idx];
	int myCol = myCell % collumns;
	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			int cell = myCell + collumns * i + j;

			if (cell < 0 || cell >= cellCount || myCol+j<0 || myCol+j >= collumns)
				continue;

			int k = gridStarts[cell];

			if (k == -1)
				continue;

			while (k<gridEnds[cell])
			{
				if (k == idx)
				{
					k++;
					continue;
				}

				float otherx = fd.devPosX[k];
				float othery = fd.devPosY[k];
				float dx = posx - otherx;
				float dy = posy - othery;


				float distsq = dx * dx + dy * dy;

				if (distsq <= sepRangesq)
				{
					float dist = sqrt(distsq);
					float d = 1 / (fmaxf(dist, 0.000001f)); //closer fish, bigger force
					sepSpeed.vx += dx*d;
					sepSpeed.vy += dy*d;

					sepNeigh = true;
				}

				if (distsq <= alignSq)
				{
					alignSpeed.vx += fd.devVelX[k];
					alignSpeed.vy += fd.devVelY[k];

					alignNeigh = true;
				}

				if (distsq <= cohSq)
				{
					xposavg += otherx;
					yposavg += othery;

					cohNeigh++;
				}
				k++;
			}

		}
	}

	if (cohNeigh > 0)
	{
		xposavg /= (float)cohNeigh;
		yposavg /= (float)cohNeigh;

		xposavg -= posx;
		yposavg -= posy;

		cohSpeed = Speed2D(xposavg, yposavg);
	}


	if (mousePos.avoid)
	{
		float mdx = posx - mousePos.x;
		float mdy = posy - mousePos.y;

		float dist = mdy * mdy + mdx * mdx;

		if (dist < 40000)
		{
			obstAvSpeed.vx += mdx;
			obstAvSpeed.vy += mdy;
			avoidingObst = true;
		}
	}

	float minSpeed = ft.minSpeed[type];
	float maxSpeed = ft.maxSpeed[type];

	alignSpeed = steerTowards(alignSpeed, vx, vy, maxSpeed, 0.2f);
	cohSpeed = steerTowards(cohSpeed, vx, vy, maxSpeed, 0.2f);
	sepSpeed = steerTowards(sepSpeed, vx, vy, maxSpeed, 0.2f);
	obstAvSpeed = steerTowards(obstAvSpeed, vx, vy, maxSpeed, 0.2f);

	Speed2D sum = Speed2D(vx, vy);

	//we add steering only when needed 
	if (alignNeigh)
		sum.addScaled(alignSpeed, alignFactor);
	if (cohNeigh)
		sum.addScaled(cohSpeed, cohFactor);
	if (sepNeigh)
		sum.addScaled(sepSpeed, sepFactor);
	if (avoidingObst)
		sum.addScaled(obstAvSpeed, obstAvFactor);

	sum.max(maxSpeed);
	sum.min(minSpeed);

	fd.devColorRGBA[idx] = ft.color[type];

	fd.devTempVelX[idx] = sum.vx;
	fd.devTempVelY[idx] = sum.vy;
}
